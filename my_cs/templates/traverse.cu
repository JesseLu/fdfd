#include "hip/hip_runtime.h"
// These macros redefine the CUDA blocks and grids to be row-major,
// instead of column major.

#define tx threadIdx.z
#define ty threadIdx.y
#define tz threadIdx.x

#define bx blockIdx.z
#define by blockIdx.y
#define bz blockIdx.x

#define txx blockDim.z
#define tyy blockDim.y
#define tzz blockDim.x

#define bxx gridDim.z
#define byy gridDim.y
#define bzz gridDim.x


// Use the complex-value definition and operators included with pycuda.
// This allows us to work with pycuda's GPUArray class.
#include <pycuda-complex.hpp>


// Defines row major access to a 3D array.
// sx, sy, sz are shifts from the present location of the field.
#define MY_OFFSET(sx,sy,sz) sx * {{ dims[1] }} * {{ dims[2] }} + sy * {{ dims[2] }} + sz 


// Macros to access fields using the field(i,j,k) format,
// where sx, sy, and sz are RELATIVE offsets in the x, y, and z directions
// respectively.
{%- for param in params%} 
#define {{ param[1] }}(sx,sy,sz) {{ param[1] }}[MY_OFFSET(sx,sy,sz)]
{%- endfor %} 

{%- for param in params%} 
#define {{ param[1] }}_abs(sx,sy,sz) {{ param[1] }}[MY_OFFSET(sx-i,sy-j,sz-k)]
{%- endfor %} 

__global__ void traverse(
    {#- Add the fields as input parameters to the function. -#}
    {%- for param in params -%} 
        {% if not loop.first -%}, {% endif -%} 
        {{ param[0] }} *{{ param[1] }}
    {%- endfor -%}) 
{
    // Set the index variables. Only i will change, since we only traverse
    // the grid in the x-direction.
    int i = tx + txx * bx;
    const int j = ty + tyy * by;
    const int k = tz + tzz * bz;

    // Set the field pointers to the appropriate location 
    // for the current thread.
    {%- for param in params %} 
    {{ param[1] }} += MY_OFFSET(i,j,k);
    {%- endfor %} 

    // User-defined "pre-loop" code.
    {{ preloop_code }}

    for (; i < {{ dims[0] }} ; i += txx) {
        // User-defined "loop" code.
        {{ loop_code }}

        // Increment the pointers, in order to scan through the entire grid
        // in the x-direction.
        {%- for param in params %} 
        {{ param[1] }} += MY_OFFSET(txx,0,0);
        {%- endfor %} 
    }

    // User-defined "post-loop" code.
    {{ postloop_code }}
}



