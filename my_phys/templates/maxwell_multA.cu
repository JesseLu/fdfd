#include "hip/hip_runtime.h"
        if ((i < {{ dims[0] }}) && (j < {{ dims[1] }}) && (k < {{ dims[2] }})) {
            int in, ip, jn, jp, kn, kp; // relative indices for adjacept points.

            if (i == 0) 
                in = {{ dims[0]-1 }};
            else
                in = -1;

            if (i == {{ dims[0]-1 }})
                ip = {{ -(dims[0]-1) }};
            else
                ip = 1;

            if (j == 0) 
                jn = {{ dims[1]-1 }};
            else
                jn = -1;

            if (j == {{ dims[1]-1 }})
                jp = {{ -(dims[1]-1) }};
            else
                jp = 1;

            if (k == 0) 
                kn = {{ dims[2]-1 }};
            else
                kn = -1;

            if (k == {{ dims[2]-1 }})
                kp = {{ -(dims[2]-1) }};
            else
                kp = 1;

            // Update equation.
            {{ type }} Hx_0 =   sz1_f[k] * (Ey(0,0,0) - Ey(0,0,kp)) - 
                                sy1_f[j] * (Ez(0,0,0) - Ez(0,jp,0));
            {{ type }} Hx_jn =  sz1_f[k] * (Ey(0,jn,0) - Ey(0,jn,kp)) - 
                                sy1_f[j+jn] * (Ez(0,jn,0) - Ez(0,0,0));
            {{ type }} Hx_kn =  sz1_f[k+kn] * (Ey(0,0,kn) - Ey(0,0,0)) - 
                                sy1_f[j] * (Ez(0,0,kn) - Ez(0,jp,kn));

            {{ type }} Hy_0 =   sx1_f[i] * (Ez(0,0,0) - Ez(ip,0,0)) - 
                                sz1_f[k] * (Ex(0,0,0) - Ex(0,0,kp));
            {{ type }} Hy_in =  sx1_f[i+in] * (Ez(in,0,0) - Ez(0,0,0)) - 
                                sz1_f[k] * (Ex(in,0,0) - Ex(in,0,kp));
            {{ type }} Hy_kn =  sx1_f[i] * (Ez(0,0,kn) - Ez(ip,0,kn)) - 
                                sz1_f[k+kn] * (Ex(0,0,kn) - Ex(0,0,0));

            {{ type }} Hz_0 =   sy1_f[j] * (Ex(0,0,0) - Ex(0,jp,0)) - 
                                sx1_f[i] * (Ey(0,0,0) - Ey(ip,0,0));
            {{ type }} Hz_in =  sy1_f[j] * (Ex(in,0,0) - Ex(in,jp,0)) - 
                                sx1_f[i+in] * (Ey(in,0,0) - Ey(0,0,0));
            {{ type }} Hz_jn =  sy1_f[j+jn] * (Ex(0,jn,0) - Ex(0,0,0)) - 
                                sx1_f[i] * (Ey(0,jn,0) - Ey(ip,jn,0));

            Ax(0,0,0) = sy0_f[j] * (Hz_0 - Hz_jn) - sz0_f[k] * (Hy_0 - Hy_kn) 
                        - {{ w2 }} * Ex(0,0,0);
            Ay(0,0,0) = sz0_f[k] * (Hx_0 - Hx_kn) - sx0_f[i] * (Hz_0 - Hz_in) 
                        - {{ w2 }} * Ey(0,0,0);
            Az(0,0,0) = sx0_f[i] * (Hy_0 - Hy_in) - sy0_f[j] * (Hx_0 - Hx_jn) 
                        - {{ w2 }} * Ez(0,0,0);
        }
