#include "hip/hip_runtime.h"
        if ((i < {{ dims[0] }}) && (j < {{ dims[1] }}) && (k < {{ dims[2] }})) {
            int in, ip, jn, jp, kn, kp; // relative indices for adjacept points.

            if (i == 0) 
                in = {{ dims[0]-1 }};
            else
                in = -1;

            if (i == {{ dims[0]-1 }})
                ip = {{ -(dims[0]-1) }};
            else
                ip = 1;

            if (j == 0) 
                jn = {{ dims[1]-1 }};
            else
                jn = -1;

            if (j == {{ dims[1]-1 }})
                jp = {{ -(dims[1]-1) }};
            else
                jp = 1;

            if (k == 0) 
                kn = {{ dims[2]-1 }};
            else
                kn = -1;

            if (k == {{ dims[2]-1 }})
                kp = {{ -(dims[2]-1) }};
            else
                kp = 1;

            {{ type }} Ex_000 = Ex(0,0,0);
            {{ type }} Ex_00p = Ex(0,0,kp);
            {{ type }} Ex_n0p = Ex(in,0,kp);
            {{ type }} Ex_n00 = Ex(in,0,0);
            {{ type }} Ex_00n = Ex(0,0,kn);
            {{ type }} Ex_0p0 = Ex(0,jp,0);
            {{ type }} Ex_np0 = Ex(in,jp,0);
            {{ type }} Ex_0n0 = Ex(0,jn,0);

            {{ type }} Ey_000 = Ey(0,0,0);
            {{ type }} Ey_00p = Ey(0,0,kp);
            {{ type }} Ey_0n0 = Ey(0,jn,0);
            {{ type }} Ey_0np = Ey(0,jn,kp);
            {{ type }} Ey_00n = Ey(0,0,kn);
            {{ type }} Ey_p00 = Ey(ip,0,0);
            {{ type }} Ey_n00 = Ey(in,0,0);
            {{ type }} Ey_pn0 = Ey(ip,jn,0);

            {{ type }} Ez_000 = Ez(0,0,0);
            {{ type }} Ez_0p0 = Ez(0,jp,0);
            {{ type }} Ez_0n0 = Ez(0,jn,0);
            {{ type }} Ez_00n = Ez(0,0,kn);
            {{ type }} Ez_0pn = Ez(0,jp,kn);
            {{ type }} Ez_p00 = Ez(ip,0,0);
            {{ type }} Ez_n00 = Ez(in,0,0);
            {{ type }} Ez_p0n = Ez(ip,0,kn);

            // Update equation.
            {{ type }} Hx_0 =   sz1_f[k] * (Ey_000 - Ey_00p) - 
                                sy1_f[j] * (Ez_000 - Ez_0p0);
            {{ type }} Hx_jn =  sz1_f[k] * (Ey_0n0 - Ey_0np) - 
                                sy1_f[j+jn] * (Ez_0n0 - Ez_000);
            {{ type }} Hx_kn =  sz1_f[k+kn] * (Ey_00n - Ey_000) - 
                                sy1_f[j] * (Ez_00n - Ez_0pn);

            {{ type }} Hy_0 =   sx1_f[i] * (Ez_000 - Ez_p00) - 
                                sz1_f[k] * (Ex_000 - Ex_00p);
            {{ type }} Hy_in =  sx1_f[i+in] * (Ez_n00 - Ez_000) - 
                                sz1_f[k] * (Ex_n00 - Ex_n0p);
            {{ type }} Hy_kn =  sx1_f[i] * (Ez_00n - Ez_p0n) - 
                                sz1_f[k+kn] * (Ex_00n - Ex_000);

            {{ type }} Hz_0 =   sy1_f[j] * (Ex_000 - Ex_0p0) - 
                                sx1_f[i] * (Ey_000 - Ey_p00);
            {{ type }} Hz_in =  sy1_f[j] * (Ex_n00 - Ex_np0) - 
                                sx1_f[i+in] * (Ey_n00 - Ey_000);
            {{ type }} Hz_jn =  sy1_f[j+jn] * (Ex_0n0 - Ex_000) - 
                                sx1_f[i] * (Ey_0n0 - Ey_pn0);

            Ax(0,0,0) = sy0_f[j] * (Hz_0 - Hz_jn) - sz0_f[k] * (Hy_0 - Hy_kn) 
                        - {{ w2 }} * Ex_000;
            Ay(0,0,0) = sz0_f[k] * (Hx_0 - Hx_kn) - sx0_f[i] * (Hz_0 - Hz_in) 
                        - {{ w2 }} * Ey_000;
            Az(0,0,0) = sx0_f[i] * (Hy_0 - Hy_in) - sy0_f[j] * (Hx_0 - Hx_jn) 
                        - {{ w2 }} * Ez_000;
        }
