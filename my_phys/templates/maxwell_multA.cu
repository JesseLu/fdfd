#include "hip/hip_runtime.h"

        int in, ip, jn, jp, kn, kp; // relative ipdices for adjacept noipts.

        if (i == 0) 
            in = {{ dims[0]-1 }};
        else
            in = -1;

        if (i == {{ dims[0]-1 }})
            ip = {{ -(dims[0]-1) }};
        else
            ip = 1;

        if (j == 0) 
            jn = {{ dims[1]-1 }};
        else
            jn = -1;

        if (j == {{ dims[1]-1 }})
            jp = {{ -(dims[1]-1) }};
        else
            jp = 1;

        if (k == 0) 
            kn = {{ dims[2]-1 }};
        else
            kn = -1;

        if (k == {{ dims[2]-1 }})
            kp = {{ -(dims[2]-1) }};
        else
            kp = 1;

        // Update equation.
        {{ type }} Hx_0 = (Ey(0,0,0) - Ey(0,0,kp)) - (Ez(0,0,0) - Ez(0,jp,0));
        {{ type }} Hx_jn = (Ey(0,jn,0) - Ey(0,jn,kp)) - (Ez(0,jn,0) - Ez(0,0,0));
        {{ type }} Hx_kn = (Ey(0,0,kn) - Ey(0,0,0)) - (Ez(0,0,kn) - Ez(0,jp,kn));
        {{ type }} Hy_0 = (Ez(0,0,0) - Ez(ip,0,0)) - (Ex(0,0,0) - Ex(0,0,kp));
        {{ type }} Hy_in = (Ez(in,0,0) - Ez(0,0,0)) - (Ex(in,0,0) - Ex(in,0,kp));
        {{ type }} Hy_kn = (Ez(0,0,kn) - Ez(ip,0,kn)) - (Ex(0,0,kn) - Ex(0,0,0));
        {{ type }} Hz_0 = (Ex(0,0,0) - Ex(0,jp,0)) - (Ey(0,0,0) - Ey(ip,0,0));
        {{ type }} Hz_in = (Ex(in,0,0) - Ex(in,jp,0)) - (Ey(in,0,0) - Ey(0,0,0));
        {{ type }} Hz_jn = (Ex(0,jn,0) - Ex(0,0,0)) - (Ey(0,jn,0) - Ey(ip,jn,0));

        Ax(0,0,0) = (Hy_0 - Hy_kn) - (Hz_0 - Hz_jn) - {{ w2 }} * Ex(0,0,0);
        Ay(0,0,0) = (Hz_0 - Hz_in) - (Hx_0 - Hx_kn) - {{ w2 }} * Ey(0,0,0);
        Az(0,0,0) = (Hx_0 - Hx_jn) - (Hy_0 - Hy_in) - {{ w2 }} * Ez(0,0,0);
