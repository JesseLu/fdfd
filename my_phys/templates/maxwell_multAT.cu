#include "hip/hip_runtime.h"
        if ((i < {{ dims[0] }}) && (j < {{ dims[1] }}) && (k < {{ dims[2] }})) {
            int in, ip, jn, jp, kn, kp; // relative indices for adjacept points.

            if (i == 0) 
                in = {{ dims[0]-1 }};
            else
                in = -1;

            if (i == {{ dims[0]-1 }})
                ip = {{ -(dims[0]-1) }};
            else
                ip = 1;

            if (j == 0) 
                jn = {{ dims[1]-1 }};
            else
                jn = -1;

            if (j == {{ dims[1]-1 }})
                jp = {{ -(dims[1]-1) }};
            else
                jp = 1;

            if (k == 0) 
                kn = {{ dims[2]-1 }};
            else
                kn = -1;

            if (k == {{ dims[2]-1 }})
                kp = {{ -(dims[2]-1) }};
            else
                kp = 1;

            {{ type }} Ex_000 = Ex(0,0,0);
            {{ type }} Ex_00p = Ex(0,0,kp);
            {{ type }} Ex_n0p = Ex(in,0,kp);
            {{ type }} Ex_n00 = Ex(in,0,0);
            {{ type }} Ex_00n = Ex(0,0,kn);
            {{ type }} Ex_0p0 = Ex(0,jp,0);
            {{ type }} Ex_np0 = Ex(in,jp,0);
            {{ type }} Ex_0n0 = Ex(0,jn,0);

            {{ type }} Ey_000 = Ey(0,0,0);
            {{ type }} Ey_00p = Ey(0,0,kp);
            {{ type }} Ey_0n0 = Ey(0,jn,0);
            {{ type }} Ey_0np = Ey(0,jn,kp);
            {{ type }} Ey_00n = Ey(0,0,kn);
            {{ type }} Ey_p00 = Ey(ip,0,0);
            {{ type }} Ey_n00 = Ey(in,0,0);
            {{ type }} Ey_pn0 = Ey(ip,jn,0);

            {{ type }} Ez_000 = Ez(0,0,0);
            {{ type }} Ez_0p0 = Ez(0,jp,0);
            {{ type }} Ez_0n0 = Ez(0,jn,0);
            {{ type }} Ez_00n = Ez(0,0,kn);
            {{ type }} Ez_0pn = Ez(0,jp,kn);
            {{ type }} Ez_p00 = Ez(ip,0,0);
            {{ type }} Ez_n00 = Ez(in,0,0);
            {{ type }} Ez_p0n = Ez(ip,0,kn);

            // Update equation.
            {{ type }} Hx_0 =   (sz0_f[k] * Ey_000 - sz0_f[k+kp] * Ey_00p) - 
                                (sy0_f[j] * Ez_000 - sy0_f[j+jp] * Ez_0p0);
            {{ type }} Hx_jn =  (sz0_f[k] * Ey_0n0 - sz0_f[k+kp] * Ey_0np) - 
                                (sy0_f[j+jn] * Ez_0n0 - sy0_f[j] * Ez_000);
            {{ type }} Hx_kn =  (sz0_f[k+kn] * Ey_00n - sz0_f[k] * Ey_000) - 
                                (sy0_f[j] * Ez_00n - sy0_f[j+jp] * Ez_0pn);

            {{ type }} Hy_0 =   (sx0_f[i] * Ez_000 - sx0_f[i+ip] * Ez_p00) - 
                                (sz0_f[k] * Ex_000 - sz0_f[k+kp] * Ex_00p);
            {{ type }} Hy_in =  (sx0_f[i+in] * Ez_n00 - sx0_f[i] * Ez_000) - 
                                (sz0_f[k] * Ex_n00 - sz0_f[k+kp] * Ex_n0p);
            {{ type }} Hy_kn =  (sx0_f[i] * Ez_00n - sx0_f[i+ip] * Ez_p0n) - 
                                (sz0_f[k+kn] * Ex_00n - sz0_f[k] * Ex_000);

            {{ type }} Hz_0 =   (sy0_f[j] * Ex_000 - sy0_f[j+jp] * Ex_0p0) - 
                                (sx0_f[i] * Ey_000 - sz0_f[i+ip] * Ey_p00);
            {{ type }} Hz_in =  (sy0_f[j] * Ex_n00 - sy0_f[j+jp] * Ex_np0) - 
                                (sx0_f[i+in] * Ey_n00 - sx0_f[i] * Ey_000);
            {{ type }} Hz_jn =  (sy0_f[j+jn] * Ex_0n0 - sy0_f[j] * Ex_000) - 
                                (sx0_f[i] * Ey_0n0 - sx0_f[i+ip] * Ey_pn0);

            Ax(0,0,0) = (sy1_f[j] * Hz_0 - sy1_f[j+jn] * Hz_jn) - 
                        (sz1_f[k] * Hy_0 - sz1_f[k+kp] * Hy_kn) - 
                        {{ w2 }} * Ex(0,0,0);
            Ay(0,0,0) = (sz1_f[k] * Hx_0 - sz1_f[k+kn] * Hx_kn) - 
                        (sx1_f[i] * Hz_0 - sx1_f[i+in] * Hz_in) - 
                        {{ w2 }} * Ey(0,0,0);
            Az(0,0,0) = (sx1_f[i] * Hy_0 - sx1_f[i+in] * Hy_in) - 
                        (sy1_f[j] * Hx_0 - sy1_f[j+jn] * Hx_jn) - 
                        {{ w2 }} * Ez(0,0,0);
        }
