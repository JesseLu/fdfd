#include "hip/hip_runtime.h"
        if ((i < {{ dims[0] }}) && (j < {{ dims[1] }}) && (k < {{ dims[2] }})) {
            int in, ip, jn, jp, kn, kp; // relative indices for adjacept points.

            if (i == 0) 
                in = {{ dims[0]-1 }};
            else
                in = -1;

            if (i == {{ dims[0]-1 }})
                ip = {{ -(dims[0]-1) }};
            else
                ip = 1;

            if (j == 0) 
                jn = {{ dims[1]-1 }};
            else
                jn = -1;

            if (j == {{ dims[1]-1 }})
                jp = {{ -(dims[1]-1) }};
            else
                jp = 1;

            if (k == 0) 
                kn = {{ dims[2]-1 }};
            else
                kn = -1;

            if (k == {{ dims[2]-1 }})
                kp = {{ -(dims[2]-1) }};
            else
                kp = 1;

            {{ type }} Ex_000 = Ex(0,0,0);
            {{ type }} Ex_00p = Ex(0,0,kp);
            {{ type }} Ex_n0p = Ex(in,0,kp);
            {{ type }} Ex_n00 = Ex(in,0,0);
            {{ type }} Ex_00n = Ex(0,0,kn);
            {{ type }} Ex_0p0 = Ex(0,jp,0);
            {{ type }} Ex_np0 = Ex(in,jp,0);
            {{ type }} Ex_0n0 = Ex(0,jn,0);

            {{ type }} Ey_000 = Ey(0,0,0);
            {{ type }} Ey_00p = Ey(0,0,kp);
            {{ type }} Ey_0n0 = Ey(0,jn,0);
            {{ type }} Ey_0np = Ey(0,jn,kp);
            {{ type }} Ey_00n = Ey(0,0,kn);
            {{ type }} Ey_p00 = Ey(ip,0,0);
            {{ type }} Ey_n00 = Ey(in,0,0);
            {{ type }} Ey_pn0 = Ey(ip,jn,0);

            {{ type }} Ez_000 = Ez(0,0,0);
            {{ type }} Ez_0p0 = Ez(0,jp,0);
            {{ type }} Ez_0n0 = Ez(0,jn,0);
            {{ type }} Ez_00n = Ez(0,0,kn);
            {{ type }} Ez_0pn = Ez(0,jp,kn);
            {{ type }} Ez_p00 = Ez(ip,0,0);
            {{ type }} Ez_n00 = Ez(in,0,0);
            {{ type }} Ez_p0n = Ez(ip,0,kn);

            // Update equation.
            {{ type }} Hx_0 =   (Ey_000 - Ey_00p) - (Ez_000 - Ez_0p0);
            {{ type }} Hx_jn =  (Ey_0n0 - Ey_0np) - (Ez_0n0 - Ez_000);
            {{ type }} Hx_kn =  (Ey_00n - Ey_000) - (Ez_00n - Ez_0pn);

            {{ type }} Hy_0 =   (Ez_000 - Ez_p00) - (Ex_000 - Ex_00p);
            {{ type }} Hy_in =  (Ez_n00 - Ez_000) - (Ex_n00 - Ex_n0p);
            {{ type }} Hy_kn =  (Ez_00n - Ez_p0n) - (Ex_00n - Ex_000);

            {{ type }} Hz_0 =   (Ex_000 - Ex_0p0) - (Ey_000 - Ey_p00);
            {{ type }} Hz_in =  (Ex_n00 - Ex_np0) - (Ey_n00 - Ey_000);
            {{ type }} Hz_jn =  (Ex_0n0 - Ex_000) - (Ey_0n0 - Ey_pn0);

            Ax(0,0,0) = (Hz_0 - Hz_jn) - (Hy_0 - Hy_kn) - {{ w2 }} * Ex(0,0,0);
            Ay(0,0,0) = (Hx_0 - Hx_kn) - (Hz_0 - Hz_in) - {{ w2 }} * Ey(0,0,0);
            Az(0,0,0) = (Hy_0 - Hy_in) - (Hx_0 - Hx_jn) - {{ w2 }} * Ez(0,0,0);
        }
