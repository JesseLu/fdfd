
#include <hip/hip_runtime.h>
// These macros redefine the CUDA blocks and grids to be row-major,
// instead of column major.

#define tx threadIdx.z
#define ty threadIdx.y
#define tz threadIdx.x

#define bx blockIdx.z
#define by blockIdx.y
#define bz blockIdx.x

#define txx blockDim.z
#define tyy blockDim.y
#define tzz blockDim.x

#define bxx gridDim.z
#define byy gridDim.y
#define bzz gridDim.x


