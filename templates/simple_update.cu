#include "hip/hip_runtime.h"
// Defines row major access to a 3D array.
#define IND(i,j,k) i * {{ dims[1] }} * {{ dims[2] }} + j * {{ dims[2] }} + k

// Macros to access fields using the field(i,j,k) format.
{%- for field in fields %} 
#define {{field}}(i,j,k) {{field}}[IND(i,j,k)]
{%- endfor %} 

__global__ void {{ function_name }}(
    {# Add the fields as input parameters to the function. #}
    {%- for field in fields -%} 
        {% if not loop.first -%}, {% endif -%} 
        {{ cuda_type }} *{{field}}
    {%- endfor -%}) 
{
    const int j = ty + tyy * by;
    const int k = tz + tzz * bz;
    for (int i = tx + txx * bx; i < {{dims[0] }} ; i += txx) {
    {{ code }}
    }
}
