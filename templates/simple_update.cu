#include "hip/hip_runtime.h"
// Make it simpler to access different field elements
#define IND(i,j,k) \
    i * {{ field_shape[1] }} * {{ field_shape[2] }} + \
    j * {{ field_shape[2] }} + \
    k

#define Ey(i,j,k) Ey[IND(i,j,k)]


__global__ void {{ function_name }}(
    {%- for field in fields -%} 
        {% if not loop.first -%}, {% endif -%} 
        {{ cuda_type }} *{{field}}
    {%- endfor -%}) 
{
    const int j = ty + tyy * by;
    const int k = tz + tzz * bz;
    for (int i = tx + txx * bx; i < {{field_shape[0] }} ; i += txx) {
    {{ code }}
    }
}
