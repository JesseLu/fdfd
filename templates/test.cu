#include "hip/hip_runtime.h"

__global__ void mult({{ type }} *x, {{ type }} *y, {{ type }} *z) {
    int i = tx + txx * bx;
    int j = ty + tyy * by;
    int k = tz + tzz * bz;
    int ind = i * {{ dims[1] }} * {{ dims[2] }} + j * {{ dims[2] }} + k;
    x[ind] = bx;
    y[ind] = by;
    z[ind] = bz;
}
