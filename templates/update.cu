#include "hip/hip_runtime.h"
// Defines row major access to a 3D array.
// sx, sy, sz are shifts from the present location of the field.
#define MY_OFFSET(sx,sy,sz) sx * {{ dims[1] }} * {{ dims[2] }} + sy * {{ dims[2] }} + sz 

// Macros to access fields using the field(i,j,k) format.
{%- for field in fields %} 
#define {{field}}(i,j,k) {{field}}[MY_OFFSET(i,j,k)]
{%- endfor %} 

__global__ void {{ function_name }}(
    {#- Add the fields as input parameters to the function. -#}
    {%- for field in fields -%} 
        {% if not loop.first -%}, {% endif -%} 
        {{ cuda_type }} *{{field}}
    {%- endfor -%}) 
{
    // Set the index variables. Only i will change, since we only traverse
    // the grid in the x-direction.
    int i = tx + txx * bx;
    const int j = ty + tyy * by;
    const int k = tz + tzz * bz;

    // Set the field pointers to the appropriate location 
    // for the current thread.
    {%- for field in fields %} 
    {{field}} += MY_OFFSET(i,j,k);
    {%- endfor %} 

    for (; i < {{ dims[0] }} ; i += txx) {
        // Begin user-defined loop code.
        {{ code }}
        // End user-defined loop code.

        // Increment the pointers, in order to scan through the entire grid
        // in the x-direction.
        {%- for field in fields %} 
        {{field}} += MY_OFFSET(1,0,0);
        {%- endfor %} 
    }
}
